#include <cassert>
#include <iostream>

#include <MatrixVectorMul.cuh>

void fillMatrix(float* matrix, int width, int height, float value) {
    for (int i = 0; i < height; ++i) {
        for (int j = 0; j < width; ++j) {
            matrix[i * width + j] = value;
        }
    }
}

int main() {
    int width = 1000, height = 1000;

    float* hostMatrix = (float*)malloc(sizeof(float) * width * height);
    float* hostVector = (float*)malloc(sizeof(float) * width);
    float* hostResult = (float*)malloc(sizeof(float) * width);

    fillMatrix(hostMatrix, width, height, 2.0f);
    
    for (int i = 0; i < width; ++i) {
        hostVector[i] = 3.0f;
    }

    float* deviceMatrix;
    float* deviceVector;
    float* deviceResult;

    hipMalloc(&deviceMatrix, sizeof(float) * width * height);
    hipMalloc(&deviceVector, sizeof(float) * width);
    hipMalloc(&deviceResult, sizeof(float) * width);

    hipMemcpy(deviceMatrix, hostMatrix, sizeof(float) * width * height, hipMemcpyHostToDevice);
    hipMemcpy(deviceVector, hostVector, sizeof(float) * width, hipMemcpyHostToDevice);

    dim3 blockSize(32, 32);
    dim3 numBlocks((height + blockSize.x - 1) / blockSize.x, (width + blockSize.y - 1) / blockSize.y);

    hipEvent_t start;
    hipEventCreate(&start);
    hipEvent_t stop;
    hipEventCreate(&stop);

    hipEventRecord(start);

    MatrixVectorMul<<<numBlocks, blockSize>>>(height, width, deviceMatrix, deviceVector, deviceResult);
    
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipMemcpy(hostResult, deviceResult, sizeof(float) * width, hipMemcpyDeviceToHost);

    for (int i = 0; i < width; ++i) {
        assert(hostResult[i] == 6000.0f);
    }

    float elapsedTime = 0.0f;
    hipEventElapsedTime(&elapsedTime, start, stop);

    std::cout << elapsedTime << '\n';

    hipFree(deviceMatrix);
    hipFree(deviceVector);
    hipFree(deviceResult);

    free(hostMatrix);
    free(hostVector);
    free(hostResult);

    return 0;

}

