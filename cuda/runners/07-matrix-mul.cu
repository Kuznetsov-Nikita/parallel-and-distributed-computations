#include <iostream>
#include <cassert>

#include <MatrixMul.cuh>

void fillMatrix(float* matrix, int width, int height, float value) {
    for (int i = 0; i < height; ++i) {
        for (int j = 0; j < width; ++j) {
            matrix[i * width + j] = value;
        }
    }
}

int main() {
    int heightA = 1000, widthA = 1000, heightB = 1000, widthB = 1000;

    float* hostA = (float*)malloc(sizeof(float) * heightA * widthA);
    float* hostB = (float*)malloc(sizeof(float) * heightB * widthB);
    float* hostResult = (float*)malloc(sizeof(float) * heightA * widthB);

    fillMatrix(hostA, widthA, heightA, 1);
    fillMatrix(hostB, widthB, heightB, 1);

    float* deviceA;
    float* deviceB;
    float* deviceResult;

    hipMalloc(&deviceA, sizeof(float) * widthA * heightA);
    hipMalloc(&deviceB, sizeof(float) * widthB * heightB);
    hipMalloc(&deviceResult, sizeof(float) * widthB * heightA);

    hipMemcpy(deviceA, hostA, sizeof(float) * widthA * heightA, hipMemcpyHostToDevice);
    hipMemcpy(deviceB, hostB, sizeof(float) * widthB * heightB, hipMemcpyHostToDevice);

    dim3 blockSize(32, 32);
    dim3 numBlocks((heightA + blockSize.x - 1) / blockSize.x, (widthB + blockSize.y - 1) / blockSize.y);

    hipEvent_t start;
    hipEventCreate(&start);
    hipEvent_t stop;
    hipEventCreate(&stop);

    hipEventRecord(start);

    MatrixMul<<<numBlocks, blockSize>>>(heightA, widthA, widthB, deviceA, deviceB, deviceResult);
    
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipMemcpy(hostResult, deviceResult, sizeof(float) * heightA * widthB, hipMemcpyDeviceToHost);

    for (int i = 0; i < heightA; ++i) {
        for (int j = 0; j < widthB; ++j) {
            assert(hostResult[i * widthB + j] == 1000.0f);
        }
    }

    float elapsedTime = 0.0f;
    hipEventElapsedTime(&elapsedTime, start, stop);

    std::cout << elapsedTime << '\n';

    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceResult);

    free(hostA);
    free(hostB);
    free(hostResult);

    return 0;
}

