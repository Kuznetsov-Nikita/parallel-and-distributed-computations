#include <cassert>
#include <iostream>

#include "KernelMul.cuh"

int main() {
    int numElements = 1 << 28;
    size_t size = numElements * sizeof(float);

    float* hostX = (float*)malloc(size);
    float* hostY = (float*)malloc(size);
    float* hostResult = (float*)malloc(size);

    for (int i = 0; i < numElements; ++i) {
        hostX[i] = 2.0f;
        hostY[i] = 3.0f;
    }

    float* deviceX;
    float* deviceY;
    float* deviceResult;

    hipMalloc(&deviceX, size);
    hipMalloc(&deviceY, size);
    hipMalloc(&deviceResult, size);

    hipMemcpy(deviceX, hostX, size, hipMemcpyHostToDevice);
    hipMemcpy(deviceY, hostY, size, hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (numElements + blockSize - 1) / blockSize;

    hipEvent_t start;
    hipEventCreate(&start);
    hipEvent_t stop;
    hipEventCreate(&stop);

    hipEventRecord(start);

    KernelMul<<<numBlocks, blockSize>>>(numElements, deviceX, deviceY, deviceResult);
    
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipMemcpy(hostResult, deviceResult, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < numElements; ++i) {
        assert(hostResult[i] == 6.0f);
    }

    float elapsedTime = 0.0f;
    hipEventElapsedTime(&elapsedTime, start, stop);

    std::cout << elapsedTime << '\n';

    hipFree(deviceX);
    hipFree(deviceY);
    hipFree(deviceResult);

    free(hostX);
    free(hostY);
    free(hostResult);

    return 0;
}
