#include "hip/hip_runtime.h"
#include <ScalarMul.cuh>

/*
 * Calculates scalar multiplication for block
 */
__global__
void ScalarMulBlock(int numElements, float* vector1, float* vector2, float *result) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    float sum = 0.0f;

    for (int i = index; i < numElements; i += stride) {
        sum += vector1[i] * vector2[i];
    }

    atomicAdd(&result[blockIdx.x], sum);
}

