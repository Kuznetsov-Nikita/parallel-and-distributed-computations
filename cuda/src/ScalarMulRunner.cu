#include "hip/hip_runtime.h"
#include <CommonKernels.cuh>
#include <ScalarMul.cuh>
#include <ScalarMulRunner.cuh>

__global__ void Reduce(float* inData, float* outData) {
    extern __shared__ float sharedData[];

    int tid = threadIdx.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    sharedData[tid] = inData[index];
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sharedData[tid] += sharedData[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
      outData[blockIdx.x] = sharedData[0];
    }
}

float ScalarMulTwoReductions(int numElements, float* vector1, float* vector2, int blockSize) {
    int numBlocks = (numElements + blockSize - 1) / blockSize;
    int numBlocksReduce = (numBlocks + blockSize - 1) / blockSize;

    float* deviceVector1;
    float* deviceVector2;
    float* deviceMulResult;
    float* deviceReduceResult;
    float* deviceResult;

    hipMalloc(&deviceVector1, sizeof(float) * numElements);
    hipMalloc(&deviceVector2, sizeof(float) * numElements);
    hipMalloc(&deviceMulResult, sizeof(float) * numElements);
    hipMalloc(&deviceReduceResult, sizeof(float) * numBlocks);
    hipMalloc(&deviceResult, sizeof(float) * numBlocksReduce);

    hipMemcpy(deviceVector1, vector1, sizeof(float) * numElements, hipMemcpyHostToDevice);
    hipMemcpy(deviceVector2, vector2, sizeof(float) * numElements, hipMemcpyHostToDevice);

    KernelMul<<<numBlocks, blockSize>>>(numElements, deviceVector1, deviceVector2, deviceMulResult);
    hipDeviceSynchronize();

    Reduce<<<numBlocks, blockSize, sizeof(float) * blockSize>>>(deviceMulResult, deviceReduceResult);
    hipDeviceSynchronize();

    Reduce<<<numBlocksReduce, blockSize, sizeof(float) * blockSize>>>(deviceReduceResult, deviceResult);

    float* result = (float*)malloc(sizeof(float) * numBlocksReduce);
    hipMemcpy(result, deviceResult, sizeof(float) * numBlocksReduce, hipMemcpyDeviceToHost);

    float sum = 0.0f;
    for (int i = 0; i < numBlocksReduce; ++i) {
        sum += result[i];
    }

    hipFree(deviceVector1);
    hipFree(deviceVector2);
    hipFree(deviceMulResult);
    hipFree(deviceReduceResult);
    hipFree(deviceResult);

    free(result);

    return sum;
}

float ScalarMulSumPlusReduction(int numElements, float* vector1, float* vector2, int blockSize) {
    int numBlocks = (numElements + blockSize - 1) / blockSize;
    int numBlocksReduce = (numBlocks + blockSize - 1) / blockSize;

    float* deviceVector1;
    float* deviceVector2;
    float* deviceMulResult;
    float* deviceResult;

    hipMalloc(&deviceVector1, sizeof(float) * numElements);
    hipMalloc(&deviceVector2, sizeof(float) * numElements);
    hipMalloc(&deviceMulResult, sizeof(float) * numBlocks);
    hipMalloc(&deviceResult, sizeof(float) * numBlocksReduce);

    hipMemcpy(deviceVector1, vector1, sizeof(float) * numElements, hipMemcpyHostToDevice);
    hipMemcpy(deviceVector2, vector2, sizeof(float) * numElements, hipMemcpyHostToDevice);

    ScalarMulBlock<<<numBlocks, blockSize>>>(numElements, deviceVector1, deviceVector2, deviceMulResult);
    hipDeviceSynchronize();

    Reduce<<<numBlocksReduce, blockSize, sizeof(float) * blockSize>>>(deviceMulResult, deviceResult);

    float* result = (float*)malloc(sizeof(float) * numBlocksReduce);
    hipMemcpy(result, deviceResult, sizeof(float) * numBlocksReduce, hipMemcpyDeviceToHost);

    float sum = 0.0f;
    for (int i = 0; i < numBlocksReduce; ++i) {
        sum += result[i];
    }

    hipFree(deviceVector1);
    hipFree(deviceVector2);
    hipFree(deviceMulResult);
    hipFree(deviceResult);

    free(result);

    return sum;
}

